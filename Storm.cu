#include "hip/hip_runtime.h"
#include "Storm.hpp"

#include "Main.hpp"

#include "hip/hip_runtime.h"
#include ""

#include <random>
#include <ctime>

#include <math.h>

using namespace std;

unsigned int			storm_phase_current = 0;
unsigned int			storm_phase_time	= 0;

vector<unsigned int>	storm_phase_start_ticks;
vector<unsigned int>	storm_phase_duration_ticks;
vector<float>			storm_phase_mapratio;

struct storm			storm_last;
struct storm			storm_current;
struct storm			storm_to;

__global__ void draw_storm_kernel(unsigned int *device_output_data, const unsigned int output_position, const unsigned int width, const unsigned int height, const unsigned int channels, const unsigned int camera_crop_x1, const unsigned int camera_crop_y1, const float camera_z, const struct storm storm_current, const struct storm storm_to, const unsigned int storm_alpha, const struct vector3<unsigned char> storm_color) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < width * height) {
		int current_x = (i % width);
		int current_y = (i / width);

		unsigned char* frame = (unsigned char*)&device_output_data[output_position];

		unsigned int storm_alpha = 50;
		if (sqrtf((camera_crop_x1 + current_x*camera_z - storm_current.x) * (camera_crop_x1+ current_x * camera_z - storm_current.x) + (camera_crop_y1 + current_y * camera_z - storm_current.y) * (camera_crop_y1 + current_y * camera_z - storm_current.y)) >= storm_current.radius) {
			frame[current_y * (width * channels) + current_x * channels] = (255 - storm_alpha)/255.0f * frame[current_y * (width * channels) + current_x * channels] + (storm_alpha/255.0f)			* storm_color[0];
			frame[current_y * (width * channels) + current_x * channels + 1] = (255 - storm_alpha) / 255.0f * frame[current_y * (width * channels) + current_x * channels + 1] + (storm_alpha / 255.0f) * storm_color[1];
			frame[current_y * (width * channels) + current_x * channels + 2] = (255 - storm_alpha) / 255.0f * frame[current_y * (width * channels) + current_x * channels + 2] + (storm_alpha / 255.0f) * storm_color[2];
		}
		unsigned int storm_circle_alpha = 150;
		if (sqrtf((camera_crop_x1 + current_x * camera_z - storm_to.x) * (camera_crop_x1 + current_x * camera_z - storm_to.x) + (camera_crop_y1 + current_y * camera_z - storm_to.y) * (camera_crop_y1 + current_y * camera_z - storm_to.y)) >= storm_to.radius-2.0f &&
			sqrtf((camera_crop_x1 + current_x * camera_z - storm_to.x) * (camera_crop_x1 + current_x * camera_z - storm_to.x) + (camera_crop_y1 + current_y * camera_z - storm_to.y) * (camera_crop_y1 + current_y * camera_z - storm_to.y)) <= storm_to.radius+2.0f
			) {
			frame[current_y * (width * channels) + current_x * channels] = (255 - storm_circle_alpha) / 255.0f * frame[current_y * (width * channels) + current_x * channels] + (storm_circle_alpha / 255.0f) * 255;
			frame[current_y * (width * channels) + current_x * channels + 1] = (255 - storm_circle_alpha) / 255.0f * frame[current_y * (width * channels) + current_x * channels + 1] + (storm_circle_alpha / 255.0f) * 255;
			frame[current_y * (width * channels) + current_x * channels + 2] = (255 - storm_circle_alpha) / 255.0f * frame[current_y * (width * channels) + current_x * channels + 2] + (storm_circle_alpha / 255.0f) * 255;
		}
	}
}

void launch_draw_storm_kernel(unsigned int* device_output_data, const unsigned int output_position, const unsigned int width, const unsigned int height, const unsigned int channels, const unsigned int camera_crop_x1, const unsigned int camera_crop_y1, const float camera_z, const struct storm storm_current, const struct storm storm_to, const unsigned int storm_alpha, const struct vector3<unsigned char> storm_color) {
	hipError_t err = hipSuccess;

	int threadsPerBlock = 256;
	int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

	draw_storm_kernel<<<blocksPerGrid, threadsPerBlock>>> (device_output_data, output_position, width, height, channels, camera_crop_x1, camera_crop_y1, camera_z, storm_current, storm_to, storm_alpha, storm_color);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed in draw_storm_kernel (error code %s)\n", hipGetErrorString(err));
	}
}

void storm_init() {
	storm_phase_start_ticks.emplace_back(1800);
	storm_phase_duration_ticks.emplace_back(1800);
	storm_phase_mapratio.emplace_back(0.75f);

	storm_phase_start_ticks.emplace_back(1800);
	storm_phase_duration_ticks.emplace_back(1800);
	storm_phase_mapratio.emplace_back(0.4f);

	storm_phase_start_ticks.emplace_back(1800);
	storm_phase_duration_ticks.emplace_back(1800);
	storm_phase_mapratio.emplace_back(0.2f);

	storm_phase_start_ticks.emplace_back(1800);
	storm_phase_duration_ticks.emplace_back(1800);
	storm_phase_mapratio.emplace_back(0.0f);

	float storm_radius = std::min(map_dimensions[0], map_dimensions[1]) * storm_phase_mapratio[0]/2.0f;
	int storm_center_max_x = (int)floorf(std::max<float>(map_dimensions[0] - storm_radius, 0.0f));
	int storm_center_max_y = (int)floorf(std::max<float>(map_dimensions[1] - storm_radius, 0.0f));
	
	storm_current.x = (unsigned int)map_dimensions[0] / 2.0f;
	storm_current.y = (unsigned int)map_dimensions[1] / 2.0f;
	storm_current.radius = floorf(std::max<float>(map_dimensions[0], map_dimensions[1]) * std::sqrtf(2)/2.0f);
	storm_last.x = storm_current.x;
	storm_last.y = storm_current.y;
	storm_last.radius = storm_current.radius;
	storm_to.x = (unsigned int) storm_radius + (rand() % (int)(storm_center_max_x - storm_radius));
	storm_to.y = (unsigned int) storm_radius + (rand() % (int)(storm_center_max_y - storm_radius));
	storm_to.radius = storm_radius;
}

void storm_next() {
	storm_phase_time++;
	if (storm_phase_time == storm_phase_start_ticks[storm_phase_current] + storm_phase_duration_ticks[storm_phase_current]) {
		if (storm_phase_current + 1 < storm_phase_start_ticks.size()) {
			storm_phase_current++;
			storm_phase_time = 0;
			storm_current = storm_to;
			storm_last = storm_current;

			float storm_radius_new = std::min(map_dimensions[0], map_dimensions[1]) * storm_phase_mapratio[storm_phase_current] / 2.0f;
			float max_dist_from_last_center = storm_last.radius - storm_radius_new;
			float rand_dist = std::rand() / (float)RAND_MAX * max_dist_from_last_center;
			float rand_angle = std::rand() / (float)RAND_MAX * 2 * std::_Pi;

			storm_to.x = (unsigned int)(storm_last.x + rand_dist * std::cosf(rand_angle));
			storm_to.y = (unsigned int)(storm_last.y + rand_dist * std::sinf(rand_angle));
			storm_to.radius = storm_radius_new;
		}
	}
	if (storm_phase_time > storm_phase_start_ticks[storm_phase_current] && storm_phase_time < storm_phase_start_ticks[storm_phase_current] + storm_phase_duration_ticks[storm_phase_current]) {
			int delta_x = (int)(((storm_phase_time - storm_phase_start_ticks[storm_phase_current]) / (float)storm_phase_duration_ticks[storm_phase_current]) * ((int)storm_to.x - (int)storm_last.x));
			int delta_y = (int)(((storm_phase_time - storm_phase_start_ticks[storm_phase_current]) / (float)storm_phase_duration_ticks[storm_phase_current]) * ((int)storm_to.y - (int)storm_last.y));
			storm_current.x = storm_last.x + delta_x;
			storm_current.y = storm_last.y + delta_y;
			storm_current.radius = storm_last.radius + ((storm_phase_time - storm_phase_start_ticks[storm_phase_current]) / (float)storm_phase_duration_ticks[storm_phase_current]) * (storm_to.radius - storm_last.radius);
	}
}