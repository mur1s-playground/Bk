#include "hip/hip_runtime.h"
#include "Pathing.hpp"

#include "Entity.hpp"
#include "Map.hpp"

#include "hip/hip_runtime.h"

#ifndef BRUTE_PATHING

unsigned int pathing_add(struct bit_field* bf_rw, struct bit_field* bf_pathing) {
	struct path p;
	p.resolution = { 400, 400 };

	p.from = { 0.0f, 0.0f };
	p.to = { 0.0f, 0.0f };

	p.pathing_x1y1 = { 0, 0 };
	p.pathing_x2y2 = { 399, 399 };

	p.path_calc_stage = -1;

	p.pathing_data = bit_field_add_bulk_zero(bf_pathing, p.resolution[0] * p.resolution[1]) + 1;

	return bit_field_add_bulk(bf_rw, (unsigned int*)&p, ceil(sizeof(struct path) / (float)sizeof(unsigned int)), sizeof(struct path)) + 1;
}

bool pathing_set(struct bit_field* bf_rw, unsigned int pathing_position, struct vector2<float> from, struct vector2<float> to) {
	struct path* p = (struct path*)&bf_rw->data[pathing_position];
	p->from = to;
	p->to = from;
	p->path_calc_stage = 0;

	int min_x = floorf(min(from[0], to[0]));
	int max_x = floorf(max(from[0], to[0]));

	int min_y = floorf(min(from[1], to[1]));
	int max_y = floorf(max(from[1], to[1]));

	if (max_x - min_x > p->resolution[0]-1 || max_y - min_y > p->resolution[1]-1) {
		printf("invalid path\n");
		return false;
	}

	int overhead_x = p->resolution[0] - 1 - (max_x - min_x);
	int overhead_y = p->resolution[1] - 1 - (max_y - min_y);

	p->pathing_x1y1 = { (unsigned int)max((min_x - (overhead_x / 2)), 0), (unsigned int)max((min_y - (overhead_y / 2)), 0) };
	p->pathing_x2y2 = { p->pathing_x1y1[0] + p->resolution[0] - 1, p->pathing_x1y1[1] + p->resolution[1] - 1 };

	//printf("p: x1 %i y1 %i x2 %i y2 %i from %f %f to %f %f\n", p->pathing_x1y1[0], p->pathing_x1y1[1], p->pathing_x2y2[0], p->pathing_x2y2[1], from[0], from[1], to[0], to[1]);

	bit_field_invalidate_bulk(bf_rw, pathing_position, ceil(sizeof(struct path) / (float)sizeof(unsigned int)));
	return true;
}

void pathing_get(struct bit_field* bf_rw, unsigned int pathing_position, struct bit_field* bf_pathing, struct bit_field* bf_map, int path_calc_stage) {
	struct path* p = (struct path*)&bf_rw->data[pathing_position];

	launch_calculate_pathing_kernel(p, bf_rw->device_data[0], pathing_position, bf_pathing->device_data[0], path_calc_stage,
		gm.map_dimensions,
		bf_map->device_data[0], gm.map_pathable_position);
}

vector2<float> pathing_get_next(struct bit_field* bf_rw, unsigned int pathing_position, struct bit_field* bf_pathing, struct vector2<float> position) {
	struct path* p = (struct path*)&bf_rw->data[pathing_position];

	float delta_x = 0.0f;
	float delta_y = 0.0f;

	vector2<int> min_rc = { 0, 0 };
	int min_val = INT_MAX;

	vector2<unsigned int> position_in_path = { (unsigned int)floorf(position[0]) - p->pathing_x1y1[0], (unsigned int)floorf(position[1]) - p->pathing_x1y1[1] };

	int* pathing_data = (int*)&bf_pathing->data[p->pathing_data];	
	for (int r = -1; r <= 1; r++) {
		for (int c = -1; c <= 1; c++) {
			if (r == 0 && c == 0) continue;
			vector2<int> cur_position_in_path = { (int)position_in_path[0] + c, (int)position_in_path[1] + r};
			if (cur_position_in_path[0] >= 0 && cur_position_in_path[0] < p->resolution[0] && cur_position_in_path[1] >= 0 && cur_position_in_path[1] < p->resolution[1]) {
				
				int cur_val = pathing_data[cur_position_in_path[1] * p->resolution[0] + cur_position_in_path[0]];
				if (cur_val < min_val && cur_val > 0) {
					min_val = cur_val;
					min_rc = { c, r };
				}
			}
		}
	}
	return vector2<float>((float)min_rc[0], (float)min_rc[1]);
}

__global__ void calculate_pathing_kernel(
		const unsigned int* device_data_rw, const unsigned int path_position, unsigned int* device_data_pathing, const int path_calc_stage,
		const struct vector2<unsigned int> map_dimensions_center,
		const unsigned int* device_data_map, const unsigned int map_pathables
	) {

		int i = blockIdx.x * blockDim.x + threadIdx.x;

		struct path* p = (struct path*)&device_data_rw[path_position];

		if (i < p->resolution[0] * p->resolution[1]) {
			int p_row = i / p->resolution[0];
			int p_col = i % p->resolution[0];

			int p_game_row = p->pathing_x1y1[1] + p_row;
			int p_game_col = p->pathing_x1y1[0] + p_col;

			int* path_d = (int*)&device_data_pathing[p->pathing_data];

			if (path_calc_stage == 0) {
				p->path_calc_stage = 0;
				unsigned char* frame_pathable = (unsigned char*)&device_data_map[map_pathables];
				path_d[p_row * p->resolution[1] + p_col] = INT_MAX;

				bool set = false;
				
				if (p_game_row >= map_dimensions_center[1] || p_game_col >= map_dimensions_center[0] || frame_pathable[(int)floorf(p_game_row) * map_dimensions_center[0] + (int)floorf(p_game_col)] == 0) {
					path_d[p_row * p->resolution[0] + p_col] = 0;
					set = true;
				} else if ((int)floorf(p->from[0]) == p_game_col && (int)floorf(p->from[1]) == p_game_row) {
					path_d[p_row * p->resolution[0] + p_col] = 1;
					set = true;
				}
				
				int lc = 0;
				while (p->path_calc_stage == 0 && !set && lc < max(p->resolution[0], p->resolution[1])) {
					int candidate_from = INT_MAX;
					int current = path_d[p_row * p->resolution[0] + p_col];
					for (int dr = -1; dr <= 1; dr++) {
						for (int dc = -1; dc <= 1; dc++) {
							if (dr == 0 && dc == 0) continue;
							if (p_game_row + dr >= p->pathing_x1y1[1] && p_game_row + dr <= p->pathing_x2y2[1] && p_game_col + dc >= p->pathing_x1y1[0] && p_game_col + dc <= p->pathing_x2y2[0]) {
								int cur_val = path_d[(p_row + dr) * p->resolution[0] + p_col + dc];
								if (cur_val < INT_MAX && cur_val > 0 && cur_val < candidate_from) {
									candidate_from = cur_val;
								}
							}
						}
					}
					if (candidate_from < INT_MAX && candidate_from < current) {
						path_d[p_row * p->resolution[0] + p_col] = candidate_from+1;
					}
					__syncthreads();
					lc++;
				}
			} else if (path_calc_stage > 0) {
				int lc = 0;
				while (path_calc_stage > 0 && lc < max(p->resolution[0], p->resolution[1])) {
					int candidate_from = INT_MAX;
					int current = path_d[p_row * p->resolution[0] + p_col];
					for (int dr = -1; dr <= 1; dr++) {
						for (int dc = -1; dc <= 1; dc++) {
							if (dr == 0 && dc == 0) continue;
							if (p_game_row + dr >= p->pathing_x1y1[1] && p_game_row + dr <= p->pathing_x2y2[1] && p_game_col + dc >= p->pathing_x1y1[0] && p_game_col + dc <= p->pathing_x2y2[0]) {
								int cur_val = path_d[(p_row + dr) * p->resolution[0] + p_col + dc];
								if (cur_val > 0 && cur_val < candidate_from) {
									candidate_from = cur_val;
								}
							}
						}
					}
					if (candidate_from < INT_MAX && candidate_from < current) {
						path_d[p_row * p->resolution[0] + p_col] = candidate_from + 1;
					}
					__syncthreads();
					lc++;
				}
			}
		}
}

void launch_calculate_pathing_kernel(const struct path *p,
	const unsigned int* device_data_rw, const unsigned int path_position, unsigned int *device_data_pathing, const int path_calc_stage,
	const struct vector2<unsigned int> map_dimensions_center,
	const unsigned int* device_data_map, const unsigned int map_pathables) {

	hipError_t err = hipSuccess;

	int threadsPerBlock = 256;
	int blocksPerGrid = (p->resolution[0]*p->resolution[1] + threadsPerBlock - 1) / threadsPerBlock;

	calculate_pathing_kernel << <blocksPerGrid, threadsPerBlock >> > (device_data_rw, path_position, device_data_pathing, path_calc_stage,
		map_dimensions_center, device_data_map, map_pathables);
		err = hipGetLastError();

		if (err != hipSuccess) {
			fprintf(stderr, "Failed in calulate_pathing_kernel (error code %s)\n", hipGetErrorString(err));
		}

		hipDeviceSynchronize();
}
#endif

#ifdef PATHING_DEBUG

#ifndef BRUTE_PATHING

__global__ void draw_gpu_pathing_kernel(
	unsigned int* device_data_rw, const unsigned int pathing_position,
	unsigned int* device_data_pathing,
	unsigned int* device_data_output, const unsigned int output_position, const unsigned int output_width, const unsigned int output_height, const unsigned int output_channels,
	const unsigned int camera_x1, const unsigned int camera_y1, const float camera_z, const unsigned int tick_counter) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	struct path* p = (struct path*)&device_data_rw[pathing_position];

	if (i < output_width*output_height) {
		int current_x = i % output_width;
		int current_y = i / output_width;

		int current_game_x = (int)floorf(camera_x1 + current_x * camera_z);
		int current_game_y = (int)floorf(camera_y1 + current_y * camera_z);
		
		if (current_game_x >= p->pathing_x1y1[0] && current_game_x < p->pathing_x2y2[0] && current_game_y >= p->pathing_x1y1[1] && current_game_y < p->pathing_x2y2[1]) {
			int* path_data = (int*)&device_data_pathing[p->pathing_data];
			
			int r = current_game_y - (int)p->pathing_x1y1[1];
			int c = current_game_x - (int)p->pathing_x1y1[0];

			unsigned char* output = (unsigned char*)&device_data_output[output_position];
			
			if (path_data[r * p->resolution[0] + c] == 1 || path_data[r * p->resolution[0] + c] == -1) {
				output[current_y * (output_width * output_channels) + current_x * output_channels + 0] = 255;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 1] = 255;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 2] = 255;
			} else if (path_data[r * p->resolution[0] + c] == INT_MAX) {

			} else if (path_data[r * p->resolution[0] + c] < 0) {
				output[current_y * (output_width * output_channels) + current_x * output_channels + 0] = 0;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 1] = 0;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 2] = (char)(-path_data[r * p->resolution[0] + c] / 3.0f);
			} else if (path_data[r * p->resolution[0] + c] > 0) {
				output[current_y * (output_width * output_channels) + current_x * output_channels + 0] = 0;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 1] = (char)(path_data[r * p->resolution[0] + c] / 3.0f);
				output[current_y * (output_width * output_channels) + current_x * output_channels + 2] = 0;
			} else if (path_data[r * p->resolution[0] + c] == 0) {
				output[current_y * (output_width * output_channels) + current_x * output_channels + 0] = 255;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 1] = 0;
				output[current_y * (output_width * output_channels) + current_x * output_channels + 2] = 0;
			}
		}
	}
}

void launch_draw_gpu_kernel(
	unsigned int* device_data_rw, const unsigned int pathing_position,
	unsigned int* device_data_pathing,
	unsigned int* device_data_output, const unsigned int output_position, const unsigned int output_width, const unsigned int output_height, const unsigned int output_channels,
	const unsigned int camera_x1, const unsigned int camera_y1, const float camera_z, const unsigned int tick_counter) {

	hipError_t err = hipSuccess;

		int threadsPerBlock = 256;
		int blocksPerGrid = (output_width*output_height + threadsPerBlock - 1) / threadsPerBlock;

		draw_gpu_pathing_kernel << <blocksPerGrid, threadsPerBlock >> > (device_data_rw, pathing_position, device_data_pathing,
			device_data_output, output_position, output_width, output_height, output_channels,
			camera_x1, camera_y1, camera_z, tick_counter);
		err = hipGetLastError();

		if (err != hipSuccess) {
			fprintf(stderr, "Failed in draw_particles_kernel (error code %s)\n", hipGetErrorString(err));
		}
	
}
#else

__global__ void draw_pathing_kernel(
	const unsigned int* device_data_assets,
	const unsigned int* device_data_rw, const unsigned int entities_position, const unsigned int entities_count,
	const struct vector2<unsigned int> map_dimensions_center,
	unsigned int* device_data_output, const unsigned int output_position, const unsigned int output_width, const unsigned int output_height, const unsigned int output_channels,
	const unsigned int camera_x1, const unsigned int camera_y1, const float camera_z, const unsigned int tick_counter) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	struct entity* entities = (struct entity*)&device_data_rw[entities_position];

	if (i < entities_count) {
		if (entities[i].et == ET_PLAYER) {
			int* params = (int*)&entities[i].params;
			int params_pos = 1;
			for (int ip = 0; ip < 6; ip++) {
				params_pos++;
				params_pos++;
			}
			int path_count = params[params_pos++];
			for (int mp = 0; mp < path_count-1; mp++) {
				int current_x = params[params_pos];
				int current_y = params[params_pos + 1];

				float current_output_x = (current_x - camera_x1) / camera_z;
				float current_output_y = (current_y - camera_y1) / camera_z;

				int current_x2 = params[params_pos + 2];
				int current_y2 = params[params_pos + 3];

				float current_output_x2 = (current_x2 - camera_x1) / camera_z;
				float current_output_y2 = (current_y2 - camera_y1) / camera_z;

				float dir_x = (current_output_x2 - current_output_x);
				float dir_y = (current_output_y2 - current_output_y);

				params_pos++;
				params_pos++;

				for (int p = 0; p < 32; p++) {
					int cur_val_x = (int)floorf(current_output_x + (p / 32.0f) * dir_x);
					int cur_val_y = (int)floorf(current_output_y + (p / 32.0f) * dir_y);
					if (cur_val_x >= 0 && cur_val_x < output_width && cur_val_y >= 0 && cur_val_y < output_height) {
						unsigned char* output = (unsigned char*)&device_data_output[output_position];
						output[cur_val_y * (output_width * output_channels) + cur_val_x * output_channels + 0] = 0;
						output[cur_val_y * (output_width * output_channels) + cur_val_x * output_channels + 1] = 0;
						output[cur_val_y * (output_width * output_channels) + cur_val_x * output_channels + 2] = 0;
					}
				}
			}
		}
	}
}

void launch_draw_pathing_kernel(
	const unsigned int* device_data_assets,
	const unsigned int* device_data_rw, const unsigned int entities_position, const unsigned int entities_count,
	unsigned int* device_data_output, const unsigned int output_position, const unsigned int output_width, const unsigned int output_height, const unsigned int output_channels,
	const unsigned int camera_x1, const unsigned int camera_y1, const float camera_z, const unsigned int tick_counter) {

	hipError_t err = hipSuccess;

	if (entities_count > 0) {

		int threadsPerBlock = 256;
		int blocksPerGrid = (entities_count + threadsPerBlock - 1) / threadsPerBlock;

		draw_pathing_kernel << <blocksPerGrid, threadsPerBlock >> > (device_data_assets,
			device_data_rw, entities_position, entities_count,
			gm.map_dimensions,
			device_data_output, output_position, output_width, output_height, output_channels,
			camera_x1, camera_y1, camera_z, tick_counter);
		err = hipGetLastError();

		if (err != hipSuccess) {
			fprintf(stderr, "Failed in draw_particles_kernel (error code %s)\n", hipGetErrorString(err));
		}
	}
}
#endif

#endif